// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "gtest/gtest.h"

#include <Core/Array/ArrayView.hpp>
#include <Core/CUDA/CUDAArray.hpp>
#include <Core/CUDA/CUDAArrayView.hpp>

using namespace CubbyFlow;

TEST(CUDAArray2, Constructors)
{
    {
        CUDAArray2<float> arr;
        EXPECT_EQ(0u, arr.Width());
        EXPECT_EQ(0u, arr.Height());
    }
    {
        CUDAArray2<float> arr(CUDAStdArray<size_t, 2>(3, 7));
        EXPECT_EQ(3u, arr.Width());
        EXPECT_EQ(7u, arr.Height());
        for (size_t i = 0; i < 21; ++i)
        {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(CUDAStdArray<size_t, 2>(1, 9), 1.5f);
        EXPECT_EQ(1u, arr.Width());
        EXPECT_EQ(9u, arr.Height());
        for (size_t i = 0; i < 9; ++i)
        {
            EXPECT_FLOAT_EQ(1.5f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(5, 2);
        EXPECT_EQ(5u, arr.Width());
        EXPECT_EQ(2u, arr.Height());
        for (size_t i = 0; i < 10; ++i)
        {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(3, 4, 7.f);
        EXPECT_EQ(3u, arr.Width());
        EXPECT_EQ(4u, arr.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ(7.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        EXPECT_EQ(4u, arr.Width());
        EXPECT_EQ(3u, arr.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr[i]);
        }
    }
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(arr);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(arr);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArrayView2<float> arrVew(arr.data(), arr.Size());
        EXPECT_EQ(4u, arrVew.Width());
        EXPECT_EQ(3u, arrVew.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arrVew[i]);
        }
    }
}

TEST(CUDAArray2, At)
{
    {
        float values[12] = { 0.f, 1.f, 2.f, 3.f, 4.f,  5.f,
                             6.f, 7.f, 8.f, 9.f, 10.f, 11.f };
        CUDAArray2<float> arr(4, 3);
        for (size_t i = 0; i < 12; ++i)
        {
            arr[i] = values[i];
        }

        // Test row-major
        EXPECT_FLOAT_EQ(0.f, arr(0, 0));
        EXPECT_FLOAT_EQ(1.f, arr(1, 0));
        EXPECT_FLOAT_EQ(2.f, arr(2, 0));
        EXPECT_FLOAT_EQ(3.f, arr(3, 0));
        EXPECT_FLOAT_EQ(4.f, arr(0, 1));
        EXPECT_FLOAT_EQ(5.f, arr(1, 1));
        EXPECT_FLOAT_EQ(6.f, arr(2, 1));
        EXPECT_FLOAT_EQ(7.f, arr(3, 1));
        EXPECT_FLOAT_EQ(8.f, arr(0, 2));
        EXPECT_FLOAT_EQ(9.f, arr(1, 2));
        EXPECT_FLOAT_EQ(10.f, arr(2, 2));
        EXPECT_FLOAT_EQ(11.f, arr(3, 2));
    }
}

TEST(CUDAArray2, CopyFrom)
{
    // From Array
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From ArrayView
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr.View());
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArray
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr.View());
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST(CUDAArray2, CopyTo)
{
    // To Array
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        Array2<float> arr2(2, 5);

        arr.CopyTo(arr2);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // To ArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        Array2<float> arr2(4, 3);
        ArrayView2<float> arrView2 = arr2.View();

        arr.CopyTo(arrView2);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArray
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr.CopyTo(arr2);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(4, 3);
        CUDAArrayView2<float> arrView2 = arr2.View();

        arr.CopyTo(arrView2);
        EXPECT_EQ(4u, arr2.Width());
        EXPECT_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST(CUDAArray2, Fill)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    arr.Fill(42.0f);
    EXPECT_EQ(4u, arr.Width());
    EXPECT_EQ(3u, arr.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ(42.0f, arr[i]);
    }
}

TEST(CUDAArray2, Resize)
{
    {
        CUDAArray2<float> arr;
        arr.Resize(CUDAStdArray<size_t, 2>(2, 9));
        EXPECT_EQ(2u, arr.Width());
        EXPECT_EQ(9u, arr.Height());
        for (size_t i = 0; i < 18; ++i)
        {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }

        arr.Resize(CUDAStdArray<size_t, 2>(8, 13), 4.f);
        hipDeviceSynchronize();
        EXPECT_EQ(8u, arr.Width());
        EXPECT_EQ(13u, arr.Height());
        for (size_t i = 0; i < 8; ++i)
        {
            for (size_t j = 0; j < 13; ++j)
            {
                if (i < 2 && j < 9)
                {
                    EXPECT_FLOAT_EQ(0.f, arr(i, j));
                }
                else
                {
                    EXPECT_FLOAT_EQ(4.f, arr(i, j));
                }
            }
        }
    }
    {
        CUDAArray2<float> arr;
        arr.Resize(7, 6);
        EXPECT_EQ(7u, arr.Width());
        EXPECT_EQ(6u, arr.Height());
        for (size_t i = 0; i < 42; ++i)
        {
            EXPECT_FLOAT_EQ(0.f, arr[i]);
        }

        arr.Resize(1, 9, 3.f);
        EXPECT_EQ(1u, arr.Width());
        EXPECT_EQ(9u, arr.Height());
        for (size_t i = 0; i < 1; ++i)
        {
            for (size_t j = 0; j < 9; ++j)
            {
                if (j < 6)
                {
                    EXPECT_FLOAT_EQ(0.f, arr(i, j));
                }
                else
                {
                    EXPECT_FLOAT_EQ(3.f, arr(i, j));
                }
            }
        }
    }
}

TEST(CUDAArray2, Clear)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    arr.Clear();
    EXPECT_EQ(0u, arr.Width());
    EXPECT_EQ(0u, arr.Height());
}

TEST(CUDAArray2, Swap)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr.Swap(arr2);

    EXPECT_EQ(2u, arr.Width());
    EXPECT_EQ(5u, arr.Height());
    for (size_t i = 0; i < 10; ++i)
    {
        EXPECT_FLOAT_EQ(42.0f, arr[i]);
    }

    EXPECT_EQ(4u, arr2.Width());
    EXPECT_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST(CUDAArray2, View)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    auto view = arr.View();

    EXPECT_EQ(4u, view.Width());
    EXPECT_EQ(3u, view.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, view[i]);
    }
    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            EXPECT_FLOAT_EQ(arr(i, j), view(i, j));
        }
    }

    const auto& arrRef = arr;
    auto constView = arrRef.View();

    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, constView[i]);
    }
    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            EXPECT_FLOAT_EQ(arr(i, j), constView(i, j));
        }
    }

    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            view(i, j) = float(i + 4 * j);
        }
    }

    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            EXPECT_FLOAT_EQ(float(i + 4 * j), arr(i, j));
            EXPECT_FLOAT_EQ(float(i + 4 * j), constView(i, j));
        }
    }
}

TEST(CUDAArray2, AssignmentOperator)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr2 = arr;

    EXPECT_EQ(4u, arr.Width());
    EXPECT_EQ(3u, arr.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr[i]);
    }

    EXPECT_EQ(4u, arr2.Width());
    EXPECT_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST(CUDAArray2, MoveOperator)
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr2 = std::move(arr);

    EXPECT_EQ(0u, arr.Width());
    EXPECT_EQ(0u, arr.Height());
    EXPECT_EQ(nullptr, arr.data());
    EXPECT_EQ(4u, arr2.Width());
    EXPECT_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        EXPECT_FLOAT_EQ((float)i + 1.f, arr2[i]);
    }
}